
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

static const unsigned int NUM_AGENTS = 1 << 15;
static const unsigned int NUM_FIRMS = 1 << 10;
static const unsigned int NUM_ITER = 1 << 20;

static const float Q = 0.2;
static const float BIAS = 0.6;
static const float P = Q + BIAS;
static const int THREADS_PER_BLOCK = 1 << 10;
static const int NUM_BLOCKS = ceil(NUM_AGENTS / THREADS_PER_BLOCK);

unsigned int *firms;
unsigned int *agents;
hiprandState *states;

    __global__
void init(unsigned int *firms, unsigned int *agents, hiprandState *states, unsigned long seed)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < NUM_AGENTS) {

        // init random states
        hiprand_init(seed, (unsigned long long)idx, 0, &states[idx]);

        // randomly select an initial firm for agent
        agents[idx] = hiprand(&states[idx]) % NUM_FIRMS;

        // tally agents assigned to firms
        atomicAdd(&firms[agents[idx]], 1);
    }
}

    __global__
void flow(unsigned int *firms, unsigned int *agents, hiprandState *states, unsigned int N)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < NUM_AGENTS) {
        hiprandState state = states[idx];
        unsigned int firm = agents[idx];
        unsigned int firmSize = firms[firm];
        unsigned int newFirm;
        float p;

        for (int i = 0; i < N; i++) {
            // randomly select another firm
            newFirm = hiprand(&state) % NUM_FIRMS;
            // compare firms to get probabiliy of moving
            p = firms[newFirm] > firmSize ? P : Q;
            if (hiprand_uniform(&state) < p) { // if moving
                // decrement tally at old firm
                atomicSub(&firms[firm], 1);
                firm = newFirm;
                // increment tally at new firm and save old tally
                firmSize = atomicAdd(&firms[firm], 1);
                // increment local tally
                firmSize++;
            }
        }
        agents[idx] = firm;
        states[idx] = state;

    }
}

/*
   __global__
   void stats(unsigned int *transactionPrice, unsigned int numTrades, unsigned int price) {
   }
 */

int main()
{
    int sum = 0;
    //unsigned long int seed = 0;
    //unsigned long int seed = (unsigned long int) time(NULL);
    unsigned long int seed = 1572534477;
    size_t firmSize = NUM_FIRMS*sizeof(unsigned int); // size of firm array
    size_t agentSize = NUM_AGENTS*sizeof(unsigned int); // size of firm array
    size_t stateSize = NUM_AGENTS*sizeof(hiprandState); // size of state array

    printf("Seed: %lu, Agents: %u, Firms: %u, Blocks: %i, Threads per block: %i, Threads: %i, Iterations: %u\n", seed, NUM_AGENTS, NUM_FIRMS, NUM_BLOCKS, THREADS_PER_BLOCK, NUM_BLOCKS * THREADS_PER_BLOCK, NUM_ITER);

    // allocate memeory
    // TODO implement error handling
    hipMallocManaged(&firms, firmSize);
    hipMallocManaged(&agents, agentSize);
    hipMallocManaged(&states, stateSize);

    init<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(firms, agents, states, seed);

    hipDeviceSynchronize();

    for (int i = 0; i < NUM_FIRMS; i++) {
        printf("%5u", firms[i]);
    }
    printf("\n");
    sum = 0;
    for (int i = 0; i < NUM_FIRMS; i++) {
        sum += firms[i];
    }
    //printf("%d\n", sum);

    for (int i = 0; i < 1; i++) {
        flow<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(firms, agents, states, NUM_ITER);
    }

    hipDeviceSynchronize();

    for (int i = 0; i < NUM_FIRMS; i++) {
        printf("%5u", firms[i]);
    }
    printf("\n");
    printf("\n");
    sum = 0;
    for (int i = 0; i < NUM_FIRMS; i++) {
        sum += firms[i];
    }
    printf("Agent Count: %d\n", sum);

    // free memory
    hipFree(firms);
    hipFree(agents);
    hipFree(states);

    return EXIT_SUCCESS;
}
