#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

static const unsigned int NUM_AGENTS = 1 << 20;
static const unsigned int MAX_FIRMS =  1 << 15;

unsigned int *firmEmployees;
hiprandState *states;

    __global__
void init(unsigned int *firmEmployees, hiprandState *states, unsigned long seed)
{
    int idx = threadIdx.x;

    // init random states
    hiprand_init(seed, idx, 0, &states[idx]);

    // distribute employees as evenly as possible
    // first disribute the evenly divisible portion
    firmEmployees[idx] = NUM_AGENTS / NUM_FIRMS;
    // then disribute the remainder
    if (idx < NUM_AGENTS % NUM_FIRMS) firmEmployees[idx] += 1;
}

    __global__
void flow(unsigned int *firmEmployees, hiprandState *states, unsigned int N)
{
    int idx = threadIdx.x;

    // activate one agent with firm-size-weighted uniform probability
    // randomly select another firm
    // decide to move or not with probabilities p and q
    // move with atomicAdd
    __syncthreads();
}

__global__
void stats(unsigned int *transactionPrice, unsigned int numTrades, unsigned int price) {
    int idx = threadIdx.x;
    unsigned int traded = 1;

    if (transactionPrice == 0) traded = 0;
}

int main()
{
    unsigned long int seed = 0;
    size_t uintSize = NUM_FIRMS*sizeof(unsigned int); // size of firm array
    size_t stateSize = NUM_BUYERS*sizeof(hiprandState); // size of state array   

    // allocate managed memeory on device
    // TODO implement error handling
    hipMallocManaged(&firmEmployees, uintSize);
    hipMallocManaged(&states, stateSize);

    init<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(firmEmployees, states, seed);

    hipDeviceSynchronize();

    flow<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(firmEmployees, N);

    hipDeviceSynchronize();

    // free memory
    hipFree(firmEmployees);

    return EXIT_SUCCESS;
}
